/*
  Programa C CUDA
  @author Juan Manuel Tortajada
  @mail ai.robotics.inbox@gmail.com
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

__global__
void operar_vectores_GPU( int n, float *A, float *B, float *C, float *D, float *E, float *F, float *G, float *H, float *K ){

  int indice_hilo_unico = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    /*
      Asegura que para una longitud de los vectores
      no multiplo del nº de hilos por bloque
      no existan hilos accediendo a posiciones de memoria fuera del vector
      (debordamiento)
    */
    if(indice_hilo_unico < n){
      C[indice_hilo_unico] = A[indice_hilo_unico] + B[indice_hilo_unico];
      F[indice_hilo_unico] = D[indice_hilo_unico] - E[indice_hilo_unico];
      G[indice_hilo_unico] = K[indice_hilo_unico] * H[indice_hilo_unico];
    }
}

int main(void){
  float tiempo_transcurrido_ms;
  hipEvent_t inicio,fin;
  hipEventCreate(&inicio);
  hipEventCreate(&fin);

  int N = 1<<20;  //  1 048 574 elementos

  float *A, *B, *C, *D, *E, *F, *G, *H, *K; //  Vectores en el Host
  float *d_A, *d_B, *d_C, *d_D, *d_E, *d_F, *d_G, *d_H, *d_K; //  Vectores en el dispositivo(GPU)

  //  Reserva de memoria en el Host

  A = (float *)malloc( N*sizeof(float) );
  B = (float *)malloc( N*sizeof(float) );
  C = (float *)malloc( N*sizeof(float) );
  D = (float *)malloc( N*sizeof(float) );
  E = (float *)malloc( N*sizeof(float) );
  F = (float *)malloc( N*sizeof(float) );
  G = (float *)malloc( N*sizeof(float) );
  H = (float *)malloc( N*sizeof(float) );
  K = (float *)malloc( N*sizeof(float) );


  //  Reserva de memoria en el dispositivo (GPU)

  hipMalloc( &d_A, N*sizeof(float) );
  hipMalloc( &d_B, N*sizeof(float) );
  hipMalloc( &d_C, N*sizeof(float) );
  hipMalloc( &d_D, N*sizeof(float) );
  hipMalloc( &d_E, N*sizeof(float) );
  hipMalloc( &d_F, N*sizeof(float) );
  hipMalloc( &d_G, N*sizeof(float) );
  hipMalloc( &d_H, N*sizeof(float) );
  hipMalloc( &d_K, N*sizeof(float) );


  //  Inicializacion de vectores (valores aleatorios [0,1e6])

  for(int i = 0; i < N; i++){
    A[i] = 1e6 * ( rand()/RAND_MAX );
    B[i] = 1e6 * ( rand()/RAND_MAX );
    C[i] = 1e6 * ( rand()/RAND_MAX );
    D[i] = 1e6 * ( rand()/RAND_MAX );
    E[i] = 1e6 * ( rand()/RAND_MAX );
    F[i] = 1e6 * ( rand()/RAND_MAX );
    G[i] = 1e6 * ( rand()/RAND_MAX );
    H[i] = 1e6 * ( rand()/RAND_MAX );
    K[i] = 1e6 * ( rand()/RAND_MAX );
  }

  //  Copia de datos del Host al Dispositivo(GPU)

  hipMemcpy( d_A, A, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_B, B, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_C, C, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_D, D, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_E, E, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_F, F, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_G, G, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_H, H, N*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_K, K, N*sizeof(float), hipMemcpyHostToDevice );


  hipEventRecord(inicio);  //  Inicio del temporizador en la GPU
  /*
    Numero de bloques: 1
    Hilos por bloque: 256
  */
  operar_vectores_GPU<<<1, 256>>>( N, d_A, d_B, d_C, d_D, d_E, d_F, d_G, d_H, d_K );
  hipEventRecord(fin);  //  Parada del temporizador en la GPU
  hipEventSynchronize(fin);  // Espera a que los datos esten listos


  //  Una vez los datos estan listos, se copia el resultado del dispositivo(GPU) al Host

  hipMemcpy( A, d_A, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( B, d_B, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( C, d_C, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( D, d_D, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( E, d_E, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( F, d_F, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( G, d_G, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( H, d_H, N*sizeof(float), hipMemcpyDeviceToHost );
  hipMemcpy( K, d_K, N*sizeof(float), hipMemcpyDeviceToHost );

  hipEventElapsedTime( &tiempo_transcurrido_ms, inicio, fin ); //  Calculo del tiempo transcurrido [ms]
  hipEventDestroy(inicio);
  hipEventDestroy(fin);

  //  Comprueba los primeros 10 elementos de los tres vectores resultado
  for(int i = 0; i < 10; i++){
    bool test1 = ( C[i] == A[i] + B[i] );
    bool test2 = ( F[i] == D[i] - E[i] );
    bool test3 = ( G[i] == K[i] * H[i] );

    printf( "\nC[%i] = A[%i] + B[%i] :%s\n", i, i, i, test1 ? "correcto" : "erroneo");
    printf( "F[%i] = D[%i] - E[%i] :%s\n", i, i, i, test2 ? "correcto" : "erroneo");
    printf( "G[%i] = K[%i] * H[%i] :%s\n", i, i, i, test3 ? "correcto" : "erroneo");

  }

  printf("\nTiempo transcurrido (GPU : kernel operarVectores) : %f ms\n\n", tiempo_transcurrido_ms);

  // Liberacion de memoria (GPU)
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);
  hipFree(d_E);
  hipFree(d_F);
  hipFree(d_G);
  hipFree(d_H);
  hipFree(d_K);

  // Liberacion de memoria (CPU)
  free(A);
  free(B);
  free(C);
  free(D);
  free(E);
  free(F);
  free(G);
  free(H);
  free(K);

  return 0;
}
